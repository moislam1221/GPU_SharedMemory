#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>
#include <time.h>

// HEADER FILES
#include "Helper/jacobi.h"
#include "Helper/residual.h"
#include "jacobi-1D-shared.h"

// IDEA: For N = 1024, create a plot of convergence time as the overlap points increase

int main(int argc, char *argv[])
{
    // INPUTS
    const int nDim = 1024; //atoi(argv[1]); 
    const int threadsPerBlock = 32; //atoi(argv[2]); 
    const float TOL = 1.0; //atoi(argv[4]);
    const int nIters = 20;
    std::string FILENAME = "OVERLAP_RESULTS/N1024_TOL1_TPB32.txt";

    // INITIALIZE ARRAYS
    int nGrids = nDim + 2;
    float * initX = new float[nGrids];
    float * rhs = new float[nGrids];
    
    // 1D POISSON MATRIX
    for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
        if (iGrid == 0 || iGrid == nGrids-1) {
            initX[iGrid] = 0.0f;
        }
        else {
            initX[iGrid] = 1.0f; 
        }
        rhs[iGrid] = 1.0f;
    }

    // NUMBER OF OVERLAP PARAMETERS TO EXPLORE
    int numIterations = (threadsPerBlock-2)/2;
    
    // NECESSARY CONTAINERS
    int * sharedCycles = new int[numIterations];
    float * sharedJacobiTimeArray = new float[numIterations];
    float * residualJacobiShared = new float[numIterations];
    float * solutionJacobiShared = new float[nGrids];
    
    // DEFINE CUDA EVENTS
    hipEvent_t start_sh, stop_sh;
    hipEventCreate(&start_sh);
    hipEventCreate(&stop_sh);
    
    int OVERLAP;
    float sharedJacobiTime;
    float totalTime = 0.0;
    for (int i = 0; i <= numIterations; i++) {
        // OBTAIN NUMBER OF CYCLES TO CONVERGE FOR GIVEN OVERLAP
        OVERLAP = 2*i;
        sharedCycles[i] = jacobiSharedIterationCount(initX, rhs, nGrids, TOL, threadsPerBlock, OVERLAP);
        for (int iter = 0; iter < nIters; iter++) {
            // GET FINAL SOLUTION
			hipEventRecord(start_sh, 0);
			solutionJacobiShared = jacobiShared(initX, rhs, nGrids, sharedCycles[i], threadsPerBlock, OVERLAP);
			// OBTAIN FINAL TIMES REQUIRED
			hipEventRecord(stop_sh, 0);
			hipEventSynchronize(stop_sh);
			hipEventElapsedTime(&sharedJacobiTime, start_sh, stop_sh);
			totalTime = totalTime + sharedJacobiTime;
            printf("FINISHED ITERATION %d\n", iter);
        }
        sharedJacobiTimeArray[i] = totalTime / nIters;
        residualJacobiShared[i] = residual1DPoisson(solutionJacobiShared, rhs, nGrids);
        printf("FINISHED OVERLAP = %d/%d CASE (N = %d, tpb = %d)\n", OVERLAP, threadsPerBlock-2, nDim, threadsPerBlock);
        totalTime = 0.0;
    }    

    // PRINTOUT
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of unknowns: %d\n", nDim);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("======================================================\n");
   
    std::ofstream timings_sh;
    timings_sh.open(FILENAME.c_str(), std::ios_base::app);
    for (int i = 0; i <= numIterations; i++) {
        int OVERLAP = 2 * i;
        printf("================================================\n");
        printf("Number of Cycles needed for Jacobi Shared for OVERLAP = %d: %d (%d) \n", OVERLAP, sharedCycles[i], threadsPerBlock/2);
        printf("Time needed for the Jacobi GPU: %f ms\n", sharedJacobiTimeArray[i]);
        printf("Residual of the Jacobi Shared solution is %f\n", residualJacobiShared[i]);
        timings_sh << OVERLAP << " " <<  sharedCycles[i] << " " << sharedJacobiTimeArray[i] << " " << residualJacobiShared[i] << "\n";
    }
    timings_sh.close(); 

    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
    delete[] solutionJacobiShared;
    
    return 0;
}
