#include "hip/hip_runtime.h"
#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>
#include <time.h>

// HEADER FILES
#include "Helper/jacobi.h"
#include "Helper/residual.h"
#include "jacobi-1D-shared.h"

// IDEA: For N = 1024, create a plot of convergence time as the overlap points increase

int main(int argc, char *argv[])
{
    // INPUTS
    const int nDim = 1024; //atoi(argv[1]); 
    const int threadsPerBlock = 32; //atoi(argv[2]); 
    const float TOL = 1.0; //atoi(argv[4]);
    const int nIters = 20;
    std::string FILENAME = "SUBITERATION_RESULTS/N1024_TOL1_TPB32.txt";

    // INITIALIZE ARRAYS
    int nGrids = nDim + 2;
    float * initX = new float[nGrids];
    float * rhs = new float[nGrids];
    
    // 1D POISSON MATRIX
    for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
        if (iGrid == 0 || iGrid == nGrids-1) {
            initX[iGrid] = 0.0f;
        }
        else {
            initX[iGrid] = 1.0f; 
        }
        rhs[iGrid] = 1.0f;
    }

    // NUMBER OF SUBITERATION AND OVERLAP PARAMETERS TO EXPLORE
    int numOverlap = threadsPerBlock/2;
    int numSubIteration = 0;
    for (int i = threadsPerBlock / 4; i < threadsPerBlock * threadsPerBlock; i = i * 2) {
        numSubIteration = numSubIteration + 1;
    }
    int numIterations = numOverlap * numSubIteration;
    
    // NECESSARY CONTAINERS
    int * sharedCycles = new int[numIterations];
    float * sharedJacobiTimeArray = new float[numIterations];
    float * residualJacobiShared = new float[numIterations];
    float * solutionJacobiShared = new float[nGrids];
    
    // DEFINE CUDA EVENTS
    hipEvent_t start_sh, stop_sh;
    hipEventCreate(&start_sh);
    hipEventCreate(&stop_sh);
    
    // PRINTOUT
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of unknowns: %d\n", nDim);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("SubIteration Values to Explore: [%d %d]\n", threadsPerBlock/4, threadsPerBlock * threadsPerBlock/2);
    printf("Overlap Range to Explore: [%d, %d]\n", 0, threadsPerBlock-2);
    printf("======================================================\n");
    
    int OVERLAP, SUBITERATIONS;
    int index;
    float sharedJacobiTime;
    float totalTime = 0.0;
    // VARY NUMBER OF SUBITERATIONS
    for (int k = 0; k < numSubIteration; k++) {
        SUBITERATIONS = (threadsPerBlock / 4) * pow(2, k);
		// VARY OVERLAP
        for (int i = 0; i < numOverlap; i++) {
			// OBTAIN NUMBER OF CYCLES TO CONVERGE FOR GIVEN COMBINATION OF OVERLAP AND SUBITERATIONS
			OVERLAP = 2*i;
            index = i + k * numOverlap;
			sharedCycles[index] = jacobiSharedIterationCount(initX, rhs, nGrids, TOL, threadsPerBlock, OVERLAP, SUBITERATIONS);
            // PERFORM 20 TRIALS
			for (int iter = 0; iter < nIters; iter++) {
				// GET FINAL SOLUTION
				hipEventRecord(start_sh, 0);
				solutionJacobiShared = jacobiShared(initX, rhs, nGrids, sharedCycles[index], threadsPerBlock, OVERLAP, SUBITERATIONS);
				// OBTAIN FINAL TIMES REQUIRED
				hipEventRecord(stop_sh, 0);
				hipEventSynchronize(stop_sh);
				hipEventElapsedTime(&sharedJacobiTime, start_sh, stop_sh);
				totalTime = totalTime + sharedJacobiTime;
				printf("FINISHED ITERATION %d\n", iter);
			}
			sharedJacobiTimeArray[index] = totalTime / nIters;
			residualJacobiShared[index] = residual1DPoisson(solutionJacobiShared, rhs, nGrids);
            printf("Residual is %f\n", residualJacobiShared[index]);
			printf("FINISHED SUBITERATIONS %d/%d OVERLAP = %d/%d CASE (N = %d, tpb = %d)\n", SUBITERATIONS, threadsPerBlock * threadsPerBlock / 2, OVERLAP, threadsPerBlock-2, nDim, threadsPerBlock);
			totalTime = 0.0;
		}    
    }
  
    // RECORD TIMINGS FOR (OVERLAP, SUBITERATIONS) IN FILE 
    std::ofstream timings_sh;
    timings_sh.open(FILENAME.c_str(), std::ios_base::app);
    for (int k = 0; k < numSubIteration; k++) {
        SUBITERATIONS = (threadsPerBlock / 4) * pow(2, k);
		for (int i = 0; i < numOverlap; i++) {
            OVERLAP = 2 * i;
            index = i + k * numOverlap;
			timings_sh << OVERLAP << " " << SUBITERATIONS << " " << sharedCycles[index] << " " << sharedJacobiTimeArray[index] << " " << residualJacobiShared[index] << "\n";
		}
    }
    timings_sh.close(); 

    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
    delete[] solutionJacobiShared;
    
    return 0;
}
