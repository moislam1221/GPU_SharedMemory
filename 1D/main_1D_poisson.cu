#include "hip/hip_runtime.h"
#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>
#include <time.h>

// HEADER FILES
#include "Helper/jacobi.h"
#include "Helper/residual.h"
#include "jacobi-1D-cpu.h"
#include "jacobi-1D-gpu.h"
#include "jacobi-1D-shared.h"

int main(int argc, char *argv[])
{
    // INPUTS
    const int nDim = 1024; //atoi(argv[1]); 
    const int threadsPerBlock = 128; //atoi(argv[2]); 
    const float TOL = 1.0; //atoi(argv[4]);
    const int OVERLAP = 0;

    // INITIALIZE ARRAYS
    int nGrids = nDim + 2;
    float * initX = new float[nGrids];
    float * rhs = new float[nGrids];
    
    // 1D POISSON MATRIX
    for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
        if (iGrid == 0 || iGrid == nGrids-1) {
            initX[iGrid] = 0.0f;
        }
        else {
            initX[iGrid] = 1.0f; 
        }
        rhs[iGrid] = 1.0f;
    }

    // OBTAIN NUMBER OF ITERATIONS NECESSARY TO ACHIEVE TOLERANCE FOR EACH METHOD
    int cpuIterations = jacobiCpuIterationCount(initX, rhs, nGrids, TOL);
    int gpuIterations = jacobiGpuIterationCount(initX, rhs, nGrids, TOL, threadsPerBlock);
    int sharedCycles = jacobiSharedIterationCount(initX, rhs, nGrids, TOL, threadsPerBlock, OVERLAP);
    
    // CPU - JACOBI
    clock_t cpuJacobiStartTime = clock();
    float * solutionJacobiCpu = jacobiCpu(initX, rhs, nGrids, cpuIterations);
    clock_t cpuJacobiEndTime = clock();
    double cpuJacobiTime = (cpuJacobiEndTime - cpuJacobiStartTime) / (float) CLOCKS_PER_SEC;
    cpuJacobiTime = cpuJacobiTime * (1e3); // Convert to ms

    // GPU - JACOBI
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float * solutionJacobiGpu = jacobiGpu(initX, rhs, nGrids, gpuIterations, threadsPerBlock);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gpuJacobiTime;
    hipEventElapsedTime(&gpuJacobiTime, start, stop);
    
    // SHARED - JACOBI
    hipEvent_t start_sh, stop_sh;
    hipEventCreate(&start_sh);
    hipEventCreate(&stop_sh);
    hipEventRecord(start_sh, 0);
    float * solutionJacobiShared = jacobiShared(initX, rhs, nGrids, sharedCycles, threadsPerBlock, OVERLAP);
    hipEventRecord(stop_sh, 0);
    hipEventSynchronize(stop_sh);
    float sharedJacobiTime;
    hipEventElapsedTime(&sharedJacobiTime, start_sh, stop_sh);
    
    // PRINT SOLUTION
    for (int i = 0; i < nGrids; i++) {
        printf("Grid %d = %f %f\n", i, solutionJacobiCpu[i], solutionJacobiGpu[i]);
    }

    // PRINTOUT
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of unknowns: %d\n", nDim);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("======================================================\n");
    
    // Print out number of iterations needed for each method
    printf("Number of Iterations needed for Jacobi CPU: %d \n", cpuIterations);
    printf("Number of Iterations needed for Jacobi GPU: %d \n", gpuIterations);
    printf("Number of Cycles needed for Jacobi Shared: %d (%d) \n", sharedCycles, threadsPerBlock/2);
    
    // Print out time for cpu, classic gpu, and swept gpu approaches
    printf("Time needed for the Jacobi CPU: %f ms\n", cpuJacobiTime);
    printf("Time needed for the Jacobi GPU: %f ms\n", gpuJacobiTime);
    printf("Time needed for the Jacobi GPU: %f ms\n", sharedJacobiTime);
    printf("======================================================\n");

    // Compute the residual of the resulting solution (|b-Ax|)
    float residualJacobiCpu = residual1DPoisson(solutionJacobiCpu, rhs, nGrids);
    float residualJacobiGpu = residual1DPoisson(solutionJacobiGpu, rhs, nGrids);
    float residualJacobiShared = residual1DPoisson(solutionJacobiShared, rhs, nGrids);
    printf("Residual of the Jacobi CPU solution is %f\n", residualJacobiCpu);
    printf("Residual of the Jacobi GPU solution is %f\n", residualJacobiGpu);
    printf("Residual of the Jacobi Shared solution is %f\n", residualJacobiShared);

    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
    delete[] solutionJacobiCpu;
    delete[] solutionJacobiGpu;
    delete[] solutionJacobiShared;
    
    return 0;
}
