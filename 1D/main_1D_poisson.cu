#include "hip/hip_runtime.h"
#include<utility>
#include<stdio.h>
#include<assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>
#include <time.h>
#include <string.h>
#include <utility>

// HEADER FILES
#include "Helper/jacobi.h"
#include "Helper/residual.h"
#include "jacobi-1D-cpu.h"
#include "jacobi-1D-gpu.h"

int main(int argc, char *argv[])
{
    // INPUTS
    const int nDim = 1024; //atoi(argv[1]); 
    const int threadsPerBlock = 32; //atoi(argv[2]); 
    const float TOL = 1.0; //atoi(argv[4]);

    // INITIALIZE ARRAYS
    int nGrids = nDim + 2;
    float * initX = new float[nGrids];
    float * rhs = new float[nGrids];
    
    // 1D POISSON MATRIX
    for (int iGrid = 0; iGrid < nGrids; ++iGrid) {
        if (iGrid == 0 || iGrid == nGrids-1) {
            initX[iGrid] = 0.0f;
        }
        else {
            initX[iGrid] = 1.0f; 
        }
        rhs[iGrid] = 1.0f;
    }

    // OBTAIN NUMBER OF ITERATIONS NECESSARY TO ACHIEVE TOLERANCE FOR EACH METHOD
    int cpuIterations = jacobiCpuIterationCount(initX, rhs, nGrids, TOL);
    int gpuIterations = jacobiGpuIterationCount(initX, rhs, nGrids, TOL, threadsPerBlock);
    
    // CPU - JACOBI
    clock_t cpuJacobiStartTime = clock();
    float * solutionJacobiCpu = jacobiCpu(initX, rhs, nGrids, cpuIterations);
    clock_t cpuJacobiEndTime = clock();
    double cpuJacobiTime = (cpuJacobiEndTime - cpuJacobiStartTime) / (float) CLOCKS_PER_SEC;
    cpuJacobiTime = cpuJacobiTime * (1e3); // Convert to ms

    // GPU - JACOBI
    float * solutionJacobiGpu = jacobiGpu(initX, rhs, nGrids, gpuIterations, threadsPerBlock);
    
    // PRINT SOLUTION
    for (int i = 0; i < nGrids; i++) {
        printf("Grid %d = %f %f\n", i, solutionJacobiCpu[i], solutionJacobiGpu[i]);
    }

    // PRINTOUT
    // Print parameters of the problem to screen
    printf("===============INFORMATION============================\n");
    printf("Number of unknowns: %d\n", nDim);
    printf("Threads Per Block: %d\n", threadsPerBlock);
    printf("======================================================\n");
    
    // Print out number of iterations needed for each method
    printf("Number of Iterations needed for Jacobi CPU: %d \n", cpuIterations);
    printf("Number of Iterations needed for Jacobi GPU: %d \n", gpuIterations);
    
    // Print out time for cpu, classic gpu, and swept gpu approaches
    printf("Time needed for the Jacobi CPU: %f ms\n", cpuJacobiTime);
    printf("======================================================\n");

    // Compute the residual of the resulting solution (|b-Ax|)
    float residualJacobiCpu = residual1DPoisson(solutionJacobiCpu, rhs, nGrids);
    float residualJacobiGpu = residual1DPoisson(solutionJacobiGpu, rhs, nGrids);
    printf("Residual of the Jacobi CPU solution is %f\n", residualJacobiCpu);
    printf("Residual of the Jacobi GPU solution is %f\n", residualJacobiGpu);

    // FREE MEMORY
    delete[] initX;
    delete[] rhs;
    delete[] solutionJacobiCpu;
    delete[] solutionJacobiGpu;
    
    return 0;
}
